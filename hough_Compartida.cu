#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    *acc = new int[rBins * degreeBins];
    memset(*acc, 0, sizeof(int) * rBins * degreeBins);
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++)
        for (int j = 0; j < h; j++)
        {
            int idx = j * w + i;
            if (pic[idx] > 0)
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;
                float theta = 0;
                for (int tIdx = 0; tIdx < degreeBins; tIdx++)
                {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++;
                    theta += radInc;
                }
            }
        }
}

__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

__global__ void GPU_HoughTranShared(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{

    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID > w * h)
        return;

    int i;
    int locID = threadIdx.x;
    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    __shared__ int localAcc[degreeBins * rBins];

    for (i = locID; i < degreeBins * rBins; i += blockDim.x)
        localAcc[i] = 0;

    __syncthreads();

    if (pic[gloID] > 0)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(localAcc + (rIdx * degreeBins + tIdx), 1);
        }
    }

    __syncthreads();

    for (i = locID; i < degreeBins * rBins; i += blockDim.x)
        atomicAdd(acc + i, localAcc[i]);
}

float calculateAverage(int *array, int size)
{
    float sum = 0;
    for (int i = 0; i < size; i++)
    {
        sum += array[i];
    }
    return sum / size;
}

float calculateStdDev(int *array, int size, float average)
{
    float variance = 0;
    for (int i = 0; i < size; i++)
    {
        variance += pow(array[i] - average, 2);
    }
    return sqrt(variance / size);
}

// Función para convertir la imagen a blanco y negro
void convertToBlackAndWhite(unsigned char *pic, int size, unsigned char threshold)
{
    for (int i = 0; i < size; i++)
    {
        pic[i] = pic[i] > threshold ? 255 : 0;
    }
}

int main(int argc, char **argv)
{
    int i;

    PGMImage inImg(argv[1]);

    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (i = 0; i < degreeBins; i++)
    {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = inImg.pixels;

    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    // Define CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    int blockNum = ceil(w * h / 256);
    GPU_HoughTranShared<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

    // Record the stop event
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipDeviceSynchronize();

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // Calcular el promedio y la desviación estándar
    const int arraySize = degreeBins * rBins;
    float average = calculateAverage(h_hough, arraySize);
    float stdDev = calculateStdDev(h_hough, arraySize, average);

    // Convertir la imagen a blanco y negro
    unsigned char threshold = 10; // Ajuste este valor según sea necesario
    convertToBlackAndWhite(inImg.pixels, w * h, threshold);

    // Crear una copia de la imagen de entrada para dibujar las líneas
    unsigned char *outputImage = new unsigned char[w * h * 3]; // 3 canales: RGB
    const int staticThreshold = 3000;                          // Static threshold set to 3000

    for (int i = 0; i < w * h; ++i)
    {
        outputImage[3 * i] = inImg.pixels[i];
        outputImage[3 * i + 1] = inImg.pixels[i];
        outputImage[3 * i + 2] = inImg.pixels[i];
    }

    // Dibujar las líneas cuyo peso es mayor que el umbral dinámico
    // Dibujar las líneas cuyo peso es mayor que el umbral estático
    for (int rIdx = 0; rIdx < rBins; rIdx++)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            if (h_hough[rIdx * degreeBins + tIdx] > staticThreshold)
            {
                float r = rIdx * rScale - rMax;
                float theta = tIdx * radInc;

                for (int x = 0; x < w; x++)
                {
                    int y = (int)((r - x * cos(theta)) / sin(theta));
                    if (y >= 0 && y < h)
                    {
                        int idx = y * w + x;
                        if (inImg.pixels[idx] > 0)
                        {
                            idx *= 3;
                            outputImage[idx] = 255;   // R
                            outputImage[idx + 1] = 0; // G
                            outputImage[idx + 2] = 0; // B
                        }
                    }
                }
            }
        }
    }

    // Guardar la imagen resultante en formato PNG
    stbi_write_png("output_image_compartida.png", w, h, 3, outputImage, w * 3);

    // Liberar la memoria utilizada
    delete[] outputImage;
    for (i = 0; i < degreeBins * rBins; i++)
    {
        if (cpuht[i] != h_hough[i])
            printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");
    printf("GPU Hough Transform tomo %f milisegundos\n", milliseconds);
    hipFree(d_in);
    hipFree(d_hough);
    free(h_hough);
    free(cpuht);
    free(pcCos);
    free(pcSin);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}