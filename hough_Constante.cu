#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "./stb_image_write.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
    *acc = new int[rBins * degreeBins];                // el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
    memset(*acc, 0, sizeof(int) * rBins * degreeBins); // init en ceros
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++)     // por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;
            if (pic[idx] > 0) // si pasa thresh, entonces lo marca
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;                       // y-coord has to be reversed
                float theta = 0;                              // actual angle
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) // add 1 to all lines in that pixel
                {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                    theta += radInc;
                }
            }
        }
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
//__constant__ float d_Cos[degreeBins];
//__constant__ float d_Sin[degreeBins];

//*****************************************************************
// TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
// TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
    // TODO calcular: int gloID = ?
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h)
        return;

    int xCent = w / 2;
    int yCent = h / 2;

    // TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    // TODO eventualmente usar memoria compartida para el acumulador

    if (pic[gloID] > 0)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            // TODO utilizar memoria constante para senos y cosenos
            // float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            // debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

    // TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
    // utilizar operaciones atomicas para seguridad
    // faltara sincronizar los hilos del bloque en algunos lados
}

// constant memory
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{

    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID > w * h)
        return;
    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    __syncthreads();

    if (pic[gloID] > 0)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {

            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;

            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

float calculateAverage(int *array, int size)
{
    float sum = 0;
    for (int i = 0; i < size; i++)
    {
        sum += array[i];
    }
    return sum / size;
}

float calculateStdDev(int *array, int size, float average)
{
    float variance = 0;
    for (int i = 0; i < size; i++)
    {
        variance += pow(array[i] - average, 2);
    }
    return sqrt(variance / size);
}

// Función para convertir la imagen a blanco y negro
void convertToBlackAndWhite(unsigned char *pic, int size, unsigned char threshold)
{
    for (int i = 0; i < size; i++)
    {
        pic[i] = pic[i] > threshold ? 255 : 0;
    }
}
//*****************************************************************
int main(int argc, char **argv)
{
    int i;

    PGMImage inImg(argv[1]);

    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (i = 0; i < degreeBins; i++)
    {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = inImg.pixels;

    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    // Define CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    int blockNum = ceil(w * h / 256);

    GPU_HoughTranConst<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

    // Record the stop event
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipDeviceSynchronize();

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // Calcular el promedio y la desviación estándar
    const int arraySize = degreeBins * rBins;
    float average = calculateAverage(h_hough, arraySize);
    float stdDev = calculateStdDev(h_hough, arraySize, average);

    // Convertir la imagen a blanco y negro
    unsigned char threshold = 10; // Ajuste este valor según sea necesario
    convertToBlackAndWhite(inImg.pixels, w * h, threshold);

    // Crear una copia de la imagen de entrada para dibujar las líneas
    unsigned char *outputImage = new unsigned char[w * h * 3]; // 3 canales: RGB
    const int staticThreshold = 3000;                          // Static threshold set to 3000

    for (int i = 0; i < w * h; ++i)
    {
        outputImage[3 * i] = inImg.pixels[i];
        outputImage[3 * i + 1] = inImg.pixels[i];
        outputImage[3 * i + 2] = inImg.pixels[i];
    }

    // Dibujar las líneas cuyo peso es mayor que el umbral dinámico
    // Dibujar las líneas cuyo peso es mayor que el umbral estático
    for (int rIdx = 0; rIdx < rBins; rIdx++)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            if (h_hough[rIdx * degreeBins + tIdx] > staticThreshold)
            {
                float r = rIdx * rScale - rMax;
                float theta = tIdx * radInc;

                for (int x = 0; x < w; x++)
                {
                    int y = (int)((r - x * cos(theta)) / sin(theta));
                    if (y >= 0 && y < h)
                    {
                        int idx = y * w + x;
                        if (inImg.pixels[idx] > 0)
                        {
                            idx *= 3;
                            outputImage[idx] = 255;   // R
                            outputImage[idx + 1] = 0; // G
                            outputImage[idx + 2] = 0; // B
                        }
                    }
                }
            }
        }
    }

    // Guardar la imagen resultante en formato PNG
    stbi_write_png("output_image_constante.png", w, h, 3, outputImage, w * 3);

    // Liberar la memoria utilizada
    delete[] outputImage;

    for (i = 0; i < degreeBins * rBins; i++)
    {
        if (cpuht[i] != h_hough[i])
            printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");
    printf("GPU Hough Transform tomo %f milisegundos\n", milliseconds);
    hipFree(d_in);
    hipFree(d_hough);
    free(h_hough);
    free(cpuht);
    free(pcCos);
    free(pcSin);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}