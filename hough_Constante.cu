#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "./stb_image_write.h"
#include <algorithm>

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
    *acc = new int[rBins * degreeBins];                // el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
    memset(*acc, 0, sizeof(int) * rBins * degreeBins); // init en ceros
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++)     // por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;
            if (pic[idx] > 0) // si pasa thresh, entonces lo marca
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;                       // y-coord has to be reversed
                float theta = 0;                              // actual angle
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) // add 1 to all lines in that pixel
                {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                    theta += radInc;
                }
            }
        }
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
//__constant__ float d_Cos[degreeBins];
//__constant__ float d_Sin[degreeBins];

//*****************************************************************
// TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
// TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
// __global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
// {
//     // TODO calcular: int gloID = ?
//     int gloID = blockIdx.x * blockDim.x + threadIdx.x;
//     if (gloID >= w * h)
//         return;

//     int xCent = w / 2;
//     int yCent = h / 2;

//     // TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
//     int xCoord = gloID % w - xCent;
//     int yCoord = yCent - gloID / w;

//     // TODO eventualmente usar memoria compartida para el acumulador

//     if (pic[gloID] > 0)
//     {
//         for (int tIdx = 0; tIdx < degreeBins; tIdx++)
//         {
//             // TODO utilizar memoria constante para senos y cosenos
//             // float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
//             float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
//             int rIdx = (r + rMax) / rScale;
//             // debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
//             atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
//         }
//     }

//     // TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
//     // utilizar operaciones atomicas para seguridad
//     // faltara sincronizar los hilos del bloque en algunos lados
// }

// Memoria Constante
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{

    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID > w * h)
        return;
    int xCent = w / 2;
    int yCent = h / 2;
    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    __syncthreads();

    if (pic[gloID] > 0)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {

            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;

            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

// Función para convertir la imagen a blanco y negro
void convertToBlackAndWhite(unsigned char *pic, int size, unsigned char threshold)
{
    for (int i = 0; i < size; i++)
    {
        pic[i] = pic[i] > threshold ? 255 : 0;
    }
}

void drawLines(unsigned char *outputImage, int w, int h, int *h_hough, float rMax, float rScale)
{
    // Define a threshold for considering a line as detected
    const int detectionThreshold = 3800;

    // Iterate through the Hough space to find lines with sufficient votes
    for (int rIdx = 0; rIdx < rBins; rIdx++)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            if (h_hough[rIdx * degreeBins + tIdx] > detectionThreshold)
            {
                // Convert Hough space coordinates back to image space
                float theta = tIdx * radInc;
                float r = rIdx * rScale - rMax;

                // Calculate the coordinates of two points on the line
                int x0 = static_cast<int>(w / 2 + r * cos(theta));
                int y0 = static_cast<int>(h / 2 - r * sin(theta));

                int x1 = static_cast<int>(x0 - (w / 2) * (-sin(theta)));
                int y1 = static_cast<int>(y0 + (h / 2) * (cos(theta)));

                // Clip the line coordinates to be within the image boundaries
                x0 = std::max(0, std::min(x0, w - 1));
                y0 = std::max(0, std::min(y0, h - 1));
                x1 = std::max(0, std::min(x1, w - 1));
                y1 = std::max(0, std::min(y1, h - 1));

                // Draw the line on the output image
                for (int i = 0; i < 2000; i++)
                {
                    int x = static_cast<int>(x0 + i * (x1 - x0) / (w / 2));
                    int y = static_cast<int>(y0 + i * (y1 - y0) / (h / 2));

                    // Ensure that the coordinates are within the image boundaries
                    if (x >= 0 && x < w && y >= 0 && y < h)
                    {
                        outputImage[3 * (y * w + x)] = 0;       // Canal rojo
                        outputImage[3 * (y * w + x) + 1] = 255; // Canal verde
                        outputImage[3 * (y * w + x) + 2] = 255; // Canal azul
                    }
                }
            }
        }
    }
    // Define a threshold for considering a line as detected
    // const int detectionThreshold = 3800;

    // Iterate through the Hough space to find lines with sufficient votes
    for (int rIdx = 0; rIdx < rBins; rIdx++)
    {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
            if (h_hough[rIdx * degreeBins + tIdx] > detectionThreshold)
            {
                // Convert Hough space coordinates back to image space
                float theta = tIdx * radInc;
                float r = rIdx * rScale - rMax;

                // Calculate the coordinates of two points on the line
                int x0 = static_cast<int>(w / 2 + r * cos(theta));
                int y0 = static_cast<int>(h / 2 - r * sin(theta));

                int x1 = static_cast<int>(x0 - (w / 2) * (-sin(theta)));
                int y1 = static_cast<int>(y0 + (h / 2) * (cos(theta)));

                // Clip the line coordinates to be within the image boundaries
                x0 = std::max(0, std::min(x0, w - 1));
                y0 = std::max(0, std::min(y0, h - 1));
                x1 = std::max(0, std::min(x1, w - 1));
                y1 = std::max(0, std::min(y1, h - 1));

                // Draw the line on the output image
                for (int i = 0; i < 2000; i++)
                {
                    int x = w - static_cast<int>(x0 + i * (x1 - x0) / (w / 2));
                    int y = static_cast<int>(y0 + i * (y1 - y0) / (h / 2));

                    // Ensure that the coordinates are within the image boundaries
                    if (x >= 0 && x < w && y >= 0 && y < h)
                    {
                        outputImage[3 * (y * w + x)] = 0;       // Canal rojo
                        outputImage[3 * (y * w + x) + 1] = 255; // Canal verde
                        outputImage[3 * (y * w + x) + 2] = 255; // Canal azul
                    }
                }
            }
        }
    }
}

int main(int argc, char **argv)
{
    int i;

    PGMImage inImg(argv[1]);

    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (i = 0; i < degreeBins; i++)
    {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = inImg.pixels;

    h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    // Define CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);

    int blockNum = ceil(w * h / 256);

    GPU_HoughTranConst<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

    // Record the stop event
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipDeviceSynchronize();

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    unsigned char threshold = 10;
    convertToBlackAndWhite(inImg.pixels, w * h, threshold);
    unsigned char *outputImage = new unsigned char[w * h * 3];
    for (int i = 0; i < w * h; ++i)
    {
        outputImage[3 * i] = inImg.pixels[i];
        outputImage[3 * i + 1] = inImg.pixels[i];
        outputImage[3 * i + 2] = inImg.pixels[i];
    }

    drawLines(outputImage, w, h, h_hough, rMax, rScale);
    // Guardar la imagen resultante en formato PNG
    stbi_write_png("output_image_constante.png", w, h, 3, outputImage, w * 3);

    // Liberar la memoria utilizada
    delete[] outputImage;

    const int tolerance = 1;

    for (int i = 0; i < degreeBins * rBins; i++)
    {
        // Calcula la diferencia absoluta entre los dos valores
        int diff = abs(cpuht[i] - h_hough[i]);

        // Verifica si la diferencia excede la tolerancia
        if (diff > tolerance)
            printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }

    printf("Done!\n");
    printf("GPU Hough Transform tomo %f milisegundos\n", milliseconds);
    hipFree(d_in);
    hipFree(d_hough);
    free(h_hough);
    free(cpuht);
    free(pcCos);
    free(pcSin);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();

    return 0;
}